#include"../include/parallel_graph.cuh"
#include"../include/serial_graph.h"
#include <cstdio>
#include <string>
#define INPUT_FILE "edgelist.txt"

long format_longput(int argc, char **argv, long *lengths)
{
    long V, E, source = 1, sink = 2;
    long a, b, cap;
    char *split_str_ptr;
    bool p_flag = false, vertex_flag = false;

    std::ofstream write_file (INPUT_FILE);
    // write_file << "test:" << argc << ", " << argv[1] << std::endl;
    // Open read file
    std::ifstream myfile (argv[1]);
    // std::cout << "file path: " << argv[1] << std::endl;
    std::string myline;
    // Read file
    if ( myfile.is_open() ) {
        while ( myfile ) {
            std::getline (myfile, myline);
            char *line = new char[myline.length() + 1];
            strcpy(line, myline.c_str());
            split_str_ptr = strtok(line," ");
            // Read split line
            for (long i = 0; split_str_ptr != NULL; i++)  
            {  
                // If its the first line comment
                if((i == 0) && (strcmp("p", split_str_ptr) == 0))
                    p_flag = true;
                else if(p_flag && i==2)
                    V = std::stoi(split_str_ptr);      
                else if(p_flag && i==3)
                {
                    E = std::stoi(split_str_ptr);
                    p_flag = false;
                }          
                else if ((i == 0) && (strcmp("a", split_str_ptr) == 0))
                    vertex_flag = true;
                else if ((i == 1) && vertex_flag)
                    a = std::stoi(split_str_ptr);
                else if ((i == 2) && vertex_flag)
                    b = std::stoi(split_str_ptr);
                else if ((i == 3) && vertex_flag)
                    cap = std::stoi(split_str_ptr);

                

                split_str_ptr = strtok(NULL, " "); 
            }
            if(vertex_flag)// && !(a == 1 || b == 2))
            {

                write_file << a << " " << b << " " << cap << '\n';
                vertex_flag = false;
            }
        }
    }
    argc = 5;
    lengths[0] = V;
    lengths[1] = E;
    lengths[2] = source;
    lengths[3] = sink;
    write_file.close();
    myfile.close();
    return argc;
}

long run_hong(int argc, long *lengths)
{
    // checking if sufficient number of arguments (4) are passed in CLI
    if(argc != 5)
    {
        printf("Invalid number of arguments passed during execution\n");
        exit(0);
    }

    // // reading the arguments passed in CLI
    // long V = atoi(argv[1]);
    // long E = atoi(argv[2]);
    // long source = atoi(argv[3]);
    // long sink = atoi(argv[4]);
    long V = lengths[0];
    long E = lengths[1];
    long source = lengths[2];
    long sink = lengths[3];

    std::cout << "V = " << V << " E = " << E << " V^2 " << V*V << std::endl;

    // declaring variables to store graph data on host as well as on CUDA device global memory 
    long *cpu_height,*gpu_height;
    long *cpu_excess_flow,*gpu_excess_flow;
    long *Excess_total;
    long *cpu_adjmtx,*gpu_adjmtx;
    long *cpu_rflowmtx,*gpu_rflowmtx;
    
    // allocating host memory
    cpu_height = (long*)malloc(V*sizeof(long));
    cpu_excess_flow = (long*)malloc(V*sizeof(long));
    cpu_adjmtx = (long*)malloc(V*V*sizeof(long));
    cpu_rflowmtx = (long*)malloc(V*V*sizeof(long));
    Excess_total = (long*)malloc(sizeof(long));

    // allocating CUDA device global memory
    hipMalloc((void**)&gpu_height,V*sizeof(long));
    hipMalloc((void**)&gpu_excess_flow,V*sizeof(long));
    hipMalloc((void**)&gpu_adjmtx,V*V*sizeof(long));
    hipMalloc((void**)&gpu_rflowmtx,V*V*sizeof(long));

    std::cout << " 1 " << std::endl;

    // readgraph
    readgraph(V,E,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx);

    std::cout << " 2 " << std::endl;
    // prlong(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);

    // time start

    // invoking the preflow function to initialise values in host
    preflow(V,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx,Excess_total);

    //prlong(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);

    // copying host data to CUDA device global memory
    hipMemcpy(gpu_height,cpu_height,V*sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(gpu_excess_flow,cpu_excess_flow,V*sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(gpu_adjmtx,cpu_adjmtx,V*V*sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(gpu_rflowmtx,cpu_rflowmtx,V*V*sizeof(long),hipMemcpyHostToDevice);

    // push_relabel()
    push_relabel(V,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx,Excess_total,gpu_height,gpu_excess_flow,gpu_adjmtx,gpu_rflowmtx);
    
    // store value from serial implementation
    long serial_check = check(V,E,source,sink);

    // prlong values from both implementations
    printf("The maximum flow value of this flow network as calculated by the parallel implementation is %ld\n",cpu_excess_flow[sink]);
    printf("The maximum flow of this flow network as calculated by the serial implementation is %ld\n",serial_check);
    
    // prlong correctness check result
    if(cpu_excess_flow[sink] == serial_check)
    {
        printf("Passed correctness check\n");
    }
    else
    {
        printf("Failed correctness check\n");
    }

    // free device memory
    hipFree(gpu_height);
    hipFree(gpu_excess_flow);
    hipFree(gpu_adjmtx);
    hipFree(gpu_rflowmtx);
    
    // free host memory
    free(cpu_height);
    free(cpu_excess_flow);
    free(cpu_adjmtx);
    free(cpu_rflowmtx);
    free(Excess_total);
    
    // return 0 and end program
    return 0;

}

int main(int argc, char **argv)
{
    long lengths[4];
    argc = format_longput(argc, argv, lengths);
    // long lengths[4] = {164922, 802681, 1, 2};
    run_hong(argc, lengths);
    // return 0 and end program
    return 0;
}