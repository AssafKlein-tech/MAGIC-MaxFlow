#include"../include/parallel_graph.cuh"

void push_relabel(long V, long source, long sink, long *cpu_height, long *cpu_excess_flow, long *cpu_adjmtx, long *cpu_rflowmtx, long *Excess_total, long *gpu_height, long *gpu_excess_flow, long *gpu_adjmtx, long *gpu_rflowmtx)
{
    /* Instead of checking for overflowing vertices(as in the sequential push relabel),
     * sum of excess flow values of sink and source are compared against Excess_total 
     * If the sum is lesser than Excess_total, 
     * it means that there is atleast one more vertex with excess flow > 0, apart from source and sink
     */

    /* declaring the mark and scan boolean arrays used in the global_relabel routine outside the while loop 
     * This is not to lose the mark values if it goes out of scope and gets redeclared in the next iteration 
     */

    bool *mark,*scanned;
    mark = (bool*)malloc(V*sizeof(bool));
    scanned = (bool*)malloc(V*sizeof(bool));

    // initialising mark values to false for all nodes
    for(long i = 0; i < V; i++)
    {
        mark[i] = false;
    }

    while((cpu_excess_flow[source] + cpu_excess_flow[sink]) < *Excess_total)
    {
        // copying height values to CUDA device global memory
        hipMemcpy(gpu_height,cpu_height,V*sizeof(long),hipMemcpyHostToDevice);

        prlongf("Invoking kernel\n");

        // invoking the push_relabel_kernel
        push_relabel_kernel<<<number_of_blocks_nodes,threads_per_block>>>(V,source,sink,gpu_height,gpu_excess_flow,gpu_adjmtx,gpu_rflowmtx);

        hipDeviceSynchronize();


        // copying height, excess flow and residual flow values from device to host memory
        hipMemcpy(cpu_height,gpu_height,V*sizeof(long),hipMemcpyDeviceToHost);
        hipMemcpy(cpu_excess_flow,gpu_excess_flow,V*sizeof(long),hipMemcpyDeviceToHost);
        hipMemcpy(cpu_rflowmtx,gpu_rflowmtx,V*V*sizeof(long),hipMemcpyDeviceToHost);

        prlongf("After invoking\n");
        //prlong(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        prlongf("Excess total : %d\n",*Excess_total);
        // perform the global_relabel routine on host
        global_relabel(V,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx,Excess_total,mark,scanned);

        prlongf("\nAfter global relabel\n");
        //prlong(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        prlongf("Excess total : %d\n",*Excess_total);
    }

}
